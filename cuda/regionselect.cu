
#include <hip/hip_runtime.h>

extern "C" __global__ void
regionselect(float* __restrict__  dst, float* __restrict__ src, int8_t* regions, int8_t region, int N) {

    int i = ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {
        dst[i] = (regions[i] == region? src[i]: 0.0f);
    }
}

